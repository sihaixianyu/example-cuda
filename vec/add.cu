
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>

using std::cout;

__global__ void vec_add(int* a, int* b, int* c, int num_elems) {
    int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (thread_id < num_elems) {
        c[thread_id] = a[thread_id] + b[thread_id];
    }
}

void verify_result(int* a, int* b, int* c, int N) {
    for (int i = 0; i < N; i++) {
        assert(c[i] == a[i] + b[i]);
        if (i >= N - 3) {
            cout << "Compute from GPU: " << c[i] << '\n';
            cout << "Ground True is: " << a[i] + b[i] << '\n';
        }
    }
}

int main() {
    constexpr auto num_elems = 1 << 26;
    size_t bytes = sizeof(int) * num_elems;

    int* host_a;
    int* host_b;
    int* host_c;

    hipHostMalloc(&host_a, bytes, hipHostMallocDefault);
    hipHostMalloc(&host_b, bytes, hipHostMallocDefault);
    hipHostMalloc(&host_c, bytes, hipHostMallocDefault);

    for (size_t i = 0; i < num_elems; i++) {
        host_a[i] = std::rand() % 100;
        host_b[i] = std::rand() % 100;
    }

    int* dev_a;
    int* dev_b;
    int* dev_c;
    hipMalloc(&dev_a, bytes);
    hipMalloc(&dev_b, bytes);
    hipMalloc(&dev_c, bytes);

    hipMemcpy(dev_a, host_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, bytes, hipMemcpyHostToDevice);

    auto num_threads = 1 << 10;
    auto num_blocks = (num_elems + num_threads - 1) / num_threads;

    vec_add<<<num_blocks, num_threads>>>(dev_a, dev_b, dev_c, num_elems);

    hipMemcpy(host_c, dev_c, bytes, hipMemcpyDeviceToHost);

    verify_result(host_a, host_b, host_c, num_elems);

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    cout << "COMPLETED SUCCESSFULLY" << '\n';

    return 0;
}
